#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <vector>
#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/distance.h>
#include <limits>
#include <cmath>

#include "scoringGpu.h"
#include "postProcess.h" // For Detection struct

// Simple IoU calculation for __device__ function
__device__ inline float calculateIoU(const cv::Rect& box1, const cv::Rect& box2) {
    int xA = max(box1.x, box2.x);
    int yA = max(box1.y, box2.y);
    int xB = min(box1.x + box1.width, box2.x + box2.width);
    int yB = min(box1.y + box1.height, box2.y + box2.height);

    // Intersection area
    int interArea = max(0, xB - xA) * max(0, yB - yA);

    // Union area
    int box1Area = box1.width * box1.height;
    int box2Area = box2.width * box2.height;
    float unionArea = static_cast<float>(box1Area + box2Area - interArea);

    // Compute IoU
    return (unionArea > 0.0f) ? static_cast<float>(interArea) / unionArea : 0.0f;
}

// GPU Kernel to calculate scores for each detection
__global__ void calculateTargetScoresGpuKernel(
    const Detection* d_detections,
    int num_detections,
    float* d_scores,
    int frame_width,
    int frame_height,
    float distance_weight       // Parameter for distance weighting
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_detections) {
        const Detection& det = d_detections[idx];
        const cv::Rect& box = det.box;

        // Calculate center of the box
        float centerX = box.x + box.width / 2.0f;
        float centerY = box.y + box.height / 2.0f;

        // Calculate distance from the center of the frame
        float frameCenterX = frame_width / 2.0f;
        float frameCenterY = frame_height / 2.0f;
        float dx = centerX - frameCenterX;
        float dy = centerY - frameCenterY;
        float distance_score = sqrtf(dx * dx + dy * dy) * distance_weight; // Apply distance weight

        // Final score (lower is better) - Only distance
        d_scores[idx] = distance_score;
    }
}

hipError_t calculateTargetScoresGpu(
    const Detection* d_detections,
    int num_detections,
    float* d_scores,
    int frame_width,
    int frame_height,
    float distance_weight_config,  // Renamed for clarity
    hipStream_t stream) {
    if (num_detections <= 0) {
        return hipSuccess; // Nothing to score
    }

    const int block_size = 256;
    const int grid_size = (num_detections + block_size - 1) / block_size;

    calculateTargetScoresGpuKernel<<<grid_size, block_size, 0, stream>>>( 
        d_detections,
        num_detections,
        d_scores,
        frame_width,
        frame_height,
        distance_weight_config       // Pass distance weight parameter
    );

    return hipGetLastError();
}

// Function to find the best target index using Thrust
hipError_t findBestTargetGpu(
    const float* d_scores,
    int num_detections,
    int* d_best_index_gpu,
    hipStream_t stream)
{
    if (num_detections <= 0) {
         // Set index to -1 (0xFFFFFFFF) if no detections
         hipMemsetAsync(d_best_index_gpu, 0xFF, sizeof(int), stream);
         return hipSuccess;
    }
    try {
        thrust::device_ptr<const float> d_scores_ptr(d_scores);

        // Use min_element because lower scores are better
        auto min_iter = thrust::min_element(
            thrust::cuda::par.on(stream),
            d_scores_ptr,
            d_scores_ptr + num_detections
        );

        // Calculate the index of the minimum element
        int best_index = thrust::distance(d_scores_ptr, min_iter);

        // Copy the best index to the output GPU buffer
        hipMemcpyAsync(
            d_best_index_gpu,
            &best_index,
            sizeof(int),
            hipMemcpyHostToDevice,
            stream
        );
        return hipGetLastError();
    } catch (const std::exception& e) {
         fprintf(stderr, "[Thrust Error] findBestTargetGpu: %s\n", e.what());
         // Set index to -1 on error
         hipMemsetAsync(d_best_index_gpu, 0xFF, sizeof(int), stream);
         return hipErrorUnknown;
    }
}
