#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <device_atomic_functions.h>

#include "filterGpu.h"
#include "postProcess.h" // Include Detection definition

// Kernel now applies both class-based ignore and optional HSV mask filter
__global__ __launch_bounds__(256, 8) void filterDetectionsByClassIdKernel(
    const Detection* __restrict__ input_detections,
    int num_input_detections,
    Detection* __restrict__ output_detections,
    int* __restrict__ output_count,
    const unsigned char* __restrict__ d_ignored_class_ids,
    int max_check_id,
    const unsigned char* __restrict__ d_hsv_mask,
    int mask_pitch,
    int min_hsv_pixels,
    bool remove_hsv_matches,
    int max_output_detections)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (; idx < num_input_detections; idx += stride) {
        // Load detection
        const Detection det = input_detections[idx]; // copy in register
        bool should_keep = true;

        // Class-based filtering
        if (det.classId >= 0 && det.classId < max_check_id && d_ignored_class_ids[det.classId]) {
            continue; // skip this detection
        }

        // HSV mask filtering (if provided)
        if (d_hsv_mask != nullptr) {
            int x0 = det.box.x;
            int y0 = det.box.y;
            int x1 = x0 + det.box.width;
            int y1 = y0 + det.box.height;
            int count = 0;
            for (int y = y0; y < y1; ++y) {
                const unsigned char* row = d_hsv_mask + y * mask_pitch;
                for (int x = x0; x < x1; ++x) {
                    if (row[x]) {
                        ++count;
                        if (remove_hsv_matches) {
                            // Remove mode: skip as soon as threshold reached
                            if (count >= min_hsv_pixels) {
                                goto skip_detection;
                            }
                        } else {
                            // Keep mode: break out when threshold reached
                            if (count >= min_hsv_pixels) {
                                y = y1; // force outer loop to end
                                break;
                            }
                        }
                    }
                }
            }
            // After scanning, if in keep mode and not enough pixels, skip
            if (!remove_hsv_matches && count < min_hsv_pixels) {
                goto skip_detection;
            }
        }

        // Passed all filters, write to output
        int write_idx = atomicAdd(output_count, 1);
        if (write_idx < max_output_detections) {
            output_detections[write_idx] = det;
        } else {
            atomicSub(output_count, 1);
        }

        continue;
    skip_detection:
        continue;
    }
}

hipError_t filterDetectionsByClassIdGpu(
    const Detection* d_input_detections,
    int num_input_detections,
    Detection* d_output_detections,
    int* d_output_count,
    const unsigned char* d_ignored_class_ids,
    int max_check_id,
    const unsigned char* d_hsv_mask,
    int mask_pitch,
    int min_hsv_pixels,
    bool remove_hsv_matches,
    int max_output_detections,
    hipStream_t stream)
{
    if (num_input_detections <= 0) {
        // No input detections, ensure output count is 0
        return hipMemsetAsync(d_output_count, 0, sizeof(int), stream);
    }

    // Reset output count
    hipError_t err = hipMemsetAsync(d_output_count, 0, sizeof(int), stream);
    if (err != hipSuccess) {
        fprintf(stderr, "[FilterGPU] Failed hipMemsetAsync on output count: %s\n", hipGetErrorString(err));
        return err;
    }

    int block_size = 256;
    int grid_size = (num_input_detections + block_size - 1) / block_size;

    filterDetectionsByClassIdKernel<<<grid_size, block_size, 0, stream>>>(
        d_input_detections,
        num_input_detections,
        d_output_detections,
        d_output_count,
        d_ignored_class_ids,
        max_check_id,
        d_hsv_mask,
        mask_pitch,
        min_hsv_pixels,
        remove_hsv_matches,
        max_output_detections);

    return hipGetLastError();
} 